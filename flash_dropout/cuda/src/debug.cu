#include "hip/hip_runtime.h"
#include <cutlass/util/device_memory.h>

#include <cute/tensor.hpp>
//
#include <cutlass/layout/matrix.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/gemm.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_fill.h>
#include <cutlass/util/tensor_view_io.h>

#include <bitset>
#include <cute/arch/copy.hpp>
#include <cute/arch/mma_sm75.hpp>
#include <cute/atom/copy_atom.hpp>
#include <cute/atom/mma_atom.hpp>

#include "matmul.cuh"

namespace ct = cute;

// template <int ThreadCount, class TileM, class TileK>
// auto make_gmem_tiled_copy() {
//     using copy_op = ct::AutoVectorizingCopyWithAssumedAlignment<128>;
//     auto copy_atom = ct::Copy_Atom<copy_op, ct::half_t>{};

//     auto tiled_copy = ct::make_tiled_copy(
//         copy_atom,
//         ct::Layout<ct::Shape<TileM, TileK>, ct::Stride<TileK, ct::_1>>{});

//     return tiled_copy;
// }

template <typename scalar_t, typename Layout>
auto host_tensor_to_ct_tensor_row_major(cutlass::HostTensor<scalar_t, Layout>& tensor,
                                        bool transpose = false) {
    auto view_engine = ct::make_gmem_ptr(tensor.device_data());
    int64_t row = tensor.extent().row();
    int64_t col = tensor.extent().column();
    int64_t stride = tensor.stride(0);

    if (std::is_same_v<Layout, cutlass::layout::RowMajor>) {
        if (transpose) {
            throw std::runtime_error("Unsupported transpose");
        } else {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(row, col),
                                                                ct::make_stride(stride, Int<1>{})));
        }
    } else if (std::is_same_v<Layout, cutlass::layout::ColumnMajor>) {
        if (transpose) {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(col, row),
                                                                ct::make_stride(stride, Int<1>{})));
        } else {
            throw std::runtime_error("Unsupported transpose");
        }
    } else {
        throw std::runtime_error("Unsupported layout");
    }
}

template <typename scalar_t, typename Layout>
auto host_tensor_to_ct_tensor_col_major(cutlass::HostTensor<scalar_t, Layout>& tensor,
                                        bool transpose = false) {
    auto view_engine = ct::make_gmem_ptr(tensor.device_data());
    int64_t row = tensor.extent().row();
    int64_t col = tensor.extent().column();
    int64_t stride = tensor.stride(0);

    if (std::is_same_v<Layout, cutlass::layout::RowMajor>) {
        if (transpose) {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(col, row),
                                                                ct::make_stride(Int<1>{}, stride)));
        } else {
            throw std::runtime_error("Unsupported transpose");
        }
    } else if (std::is_same_v<Layout, cutlass::layout::ColumnMajor>) {
        if (transpose) {
            throw std::runtime_error("Unsupported transpose");
        } else {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(row, col),
                                                                ct::make_stride(Int<1>{}, stride)));
        }
    } else {
        throw std::runtime_error("Unsupported layout");
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " M N K" << std::endl;
        return 1;
    }
    int64_t M = std::atoi(argv[1]);
    int64_t N = std::atoi(argv[2]);
    int64_t K = std::atoi(argv[3]);

    using scalar_t = ct::half_t;
    using layout_A = cutlass::layout::ColumnMajor;
    using layout_B = cutlass::layout::RowMajor;
    using layout_C = cutlass::layout::RowMajor;
    using layout_Mask = cutlass::layout::PackedVectorLayout;
    using Coord = layout_Mask::TensorCoord;
    using KernelTraits =
        KernelTraits<scalar_t, 64, 64, 32, 2, std::is_same_v<layout_A, cutlass::layout::RowMajor>,
                     std::is_same_v<layout_B, cutlass::layout::ColumnMajor>>;

    int64_t BLK_M = KernelTraits::BLK_M;
    int64_t BLK_K = KernelTraits::BLK_K;

    cutlass::HostTensor<scalar_t, layout_A> A({M, K});
    cutlass::HostTensor<scalar_t, layout_B> B({K, N});
    cutlass::HostTensor<scalar_t, layout_C> C({M, N});
    cutlass::HostTensor<scalar_t, layout_C> C_ref({M, N});

    cutlass::reference::host::TensorFillRandomGaussian(A.host_view(), 0);
    cutlass::reference::host::TensorFillRandomGaussian(B.host_view(), 0);
    cutlass::reference::host::TensorFill(C.host_view(), ct::half_t(0));
    cutlass::reference::host::TensorFill(C_ref.host_view(), ct::half_t(0));

    cutlass::DeviceAllocation<ct::uint128_t> mask_data(M / BLK_M);
    auto [mask, mask_T, mask_table] = make_mask(M / BLK_M, K / BLK_K, 0.5);
    mask_data.copy_from_host(mask.data());

    std::cout << "mask: " << std::endl;
    for (int64_t i = 0; i < M / BLK_M; ++i) {
        auto m = mask[i];
        std::cout << std::bitset<64>(m.hilo_.hi) << std::bitset<64>(m.hilo_.lo) << std::endl;
    }
    std::cout << "mask_T: " << std::endl;
    for (int64_t i = 0; i < K / BLK_K; ++i) {
        auto m = mask_T[i];
        std::cout << std::bitset<64>(m.hilo_.hi) << std::bitset<64>(m.hilo_.lo) << std::endl;
    }

    A.sync_device();
    B.sync_device();
    C.sync_device();

    auto A_ct = host_tensor_to_ct_tensor_col_major(A);
    auto B_ct = host_tensor_to_ct_tensor_col_major(B, true);
    auto C_ct = host_tensor_to_ct_tensor_row_major(C);
    auto mask_ct = ct::make_tensor(ct::make_gmem_ptr(mask_data.get()), ct::make_shape(M / BLK_M));

    std::cout << "A layout: " << A_ct.layout() << std::endl;
    std::cout << "B layout: " << B_ct.layout() << std::endl;
    std::cout << "C layout: " << C_ct.layout() << std::endl;
    std::cout << "mask layout: " << mask_ct.layout() << std::endl;

    cutlass::reference::host::Gemm<scalar_t, layout_A, scalar_t, layout_B, scalar_t, layout_C,
                                   ct::half_t, float>
        reference_gemm;
    reference_gemm({int(M), int(N), int(K)}, ct::half_t(1), A.host_ref(), B.host_ref(),
                   ct::half_t(0), C_ref.host_ref());
    // std::cout << "C_ref" << std::endl << C_ref.host_view() << std::endl;

    matmul<KernelTraits>(A_ct, B_ct, C_ct, mask_ct);

    C.sync_host();
    // std::cout << "C" << std::endl << C.host_view() << std::endl;

    float max_rel_err = 0.0f;
    float max_abs_err = 0.0f;

    // Find the max diff
    for (int64_t i = 0; i < M; ++i) {
        for (int64_t j = 0; j < N; ++j) {
            float c = C.host_ref().at({i, j});
            float c_ref = C_ref.host_ref().at({i, j});
            float diff = std::abs(c - c_ref);
            float rel = diff / std::abs(c_ref);
            max_abs_err = std::max(max_abs_err, diff);
            max_rel_err = std::max(max_rel_err, rel);
        }
    }
    std::cout << "Max abs err: " << max_abs_err << std::endl;
    std::cout << "Max rel err: " << max_rel_err * 100 << "%" << std::endl;

    return 0;
}
