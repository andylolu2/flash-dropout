#include "hip/hip_runtime.h"
#include <cutlass/util/device_memory.h>

#include <cute/tensor.hpp>
//
#include <cutlass/layout/matrix.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/gemm.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_fill.h>
#include <cutlass/util/tensor_view_io.h>

#include <cute/arch/copy.hpp>
#include <cute/arch/mma_sm75.hpp>
#include <cute/atom/copy_atom.hpp>
#include <cute/atom/mma_atom.hpp>

#include "matmul.cuh"

namespace ct = cute;

// template <int ThreadCount, class TileM, class TileK>
// auto make_gmem_tiled_copy() {
//     using copy_op = ct::AutoVectorizingCopyWithAssumedAlignment<128>;
//     auto copy_atom = ct::Copy_Atom<copy_op, ct::half_t>{};

//     auto tiled_copy = ct::make_tiled_copy(
//         copy_atom,
//         ct::Layout<ct::Shape<TileM, TileK>, ct::Stride<TileK, ct::_1>>{});

//     return tiled_copy;
// }

template <typename scalar_t, typename Layout>
auto host_tensor_to_ct_tensor(cutlass::HostTensor<scalar_t, Layout>& tensor,
                              bool transpose = false) {
    auto view_engine = ct::make_gmem_ptr(tensor.device_data());
    int64_t row = tensor.extent().row();
    int64_t col = tensor.extent().column();
    int64_t stride = tensor.stride(0);

    if (std::is_same_v<Layout, cutlass::layout::RowMajor>) {
        if (transpose) {
            throw std::runtime_error("Unsupported layout");
            // return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(col, row),
            //                                                     ct::make_stride(1L, stride)));
        } else {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(row, col),
                                                                ct::make_stride(stride, Int<1>{})));
        }
    } else if (std::is_same_v<Layout, cutlass::layout::ColumnMajor>) {
        if (transpose) {
            return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(col, row),
                                                                ct::make_stride(stride, Int<1>{})));
        } else {
            throw std::runtime_error("Unsupported layout");
            // return ct::make_tensor(view_engine, ct::make_layout(ct::make_shape(row, col),
            // ct::make_stride(1L, stride)));
        }
    } else {
        throw std::runtime_error("Unsupported layout");
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " M N K" << std::endl;
        return 1;
    }
    int64_t M = std::atoi(argv[1]);
    int64_t N = std::atoi(argv[2]);
    int64_t K = std::atoi(argv[3]);

    using scalar_t = ct::half_t;
    using layout_A = cutlass::layout::RowMajor;
    using layout_B = cutlass::layout::ColumnMajor;
    using layout_C = cutlass::layout::RowMajor;

    cutlass::HostTensor<scalar_t, layout_A> A({M, K});
    cutlass::HostTensor<scalar_t, layout_B> B({K, N});
    cutlass::HostTensor<scalar_t, layout_C> C({M, N});
    cutlass::HostTensor<scalar_t, layout_C> C_ref({M, N});

    // for (int i = 0; i < M; ++i) {
    //     for (int j = 0; j < K; ++j) {
    //         A.host_ref().at({i, j}) = scalar_t(float(i * K + j) / 100.0f);
    //     }
    // }
    // for (int i = 0; i < K; ++i) {
    //     for (int j = 0; j < N; ++j) {
    //         B.host_ref().at({i, j}) = scalar_t(-float(i * K + j) / 100.0f);
    //     }
    // }

    cutlass::reference::host::TensorFillRandomGaussian(A.host_view(), 0);
    cutlass::reference::host::TensorFillRandomGaussian(B.host_view(), 0);
    cutlass::reference::host::TensorFill(C.host_view(), ct::half_t(0));
    cutlass::reference::host::TensorFill(C_ref.host_view(), ct::half_t(0));
    A.sync_device();
    B.sync_device();
    C.sync_device();

    auto A_ct = host_tensor_to_ct_tensor(A);
    auto B_ct = host_tensor_to_ct_tensor(B, true);
    auto C_ct = host_tensor_to_ct_tensor(C);

    std::cout << "A layout: " << A_ct.layout() << std::endl;
    std::cout << "B layout: " << B_ct.layout() << std::endl;
    std::cout << "C layout: " << C_ct.layout() << std::endl;

    cutlass::reference::host::Gemm<scalar_t, layout_A, scalar_t, layout_B, scalar_t, layout_C,
                                   ct::half_t, float>
        reference_gemm;
    reference_gemm({int(M), int(N), int(K)}, ct::half_t(1), A.host_ref(), B.host_ref(),
                   ct::half_t(0), C_ref.host_ref());
    // std::cout << "C_ref" << std::endl << C_ref.host_view() << std::endl;

    matmul(A_ct, B_ct, C_ct);

    C.sync_host();
    // std::cout << "C" << std::endl << C.host_view() << std::endl;

    float max_rel_err = 0.0f;
    float max_abs_err = 0.0f;

    // Find the max diff
    for (int64_t i = 0; i < M; ++i) {
        for (int64_t j = 0; j < N; ++j) {
            float c = C.host_ref().at({i, j});
            float c_ref = C_ref.host_ref().at({i, j});
            float diff = std::abs(c - c_ref);
            float rel = diff / std::abs(c_ref);
            max_abs_err = std::max(max_abs_err, diff);
            max_rel_err = std::max(max_rel_err, rel);
        }
    }
    std::cout << "Max abs err: " << max_abs_err << std::endl;
    std::cout << "Max rel err: " << max_rel_err * 100 << "%" << std::endl;

    return 0;
}
