#include "hip/hip_runtime.h"
#include <cutlass/util/device_memory.h>

#include <cute/tensor.hpp>
//
#include <cute/arch/copy.hpp>
#include <cute/arch/mma_sm75.hpp>
#include <cute/atom/copy_atom.hpp>
#include <cute/atom/mma_atom.hpp>
#include <cutlass/util/GPU_Clock.hpp>

#include "matmul.cuh"

namespace ct = cute;

int main(int argc, char *argv[]) {
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " M N K iter" << std::endl;
        return 1;
    }
    int64_t M = std::atoi(argv[1]);
    int64_t N = std::atoi(argv[2]);
    int64_t K = std::atoi(argv[3]);
    size_t n_repeats = std::atoi(argv[4]);

    using scalar_t = ct::half_t;
    using LayoutA = ct::GenRowMajor;
    using LayoutB = ct::GenRowMajor;
    using KernelTraits =
        KernelTraits<scalar_t, 64, 64, 32, 2, std::is_same_v<LayoutA, ct::GenRowMajor>,
                     std::is_same_v<LayoutB, ct::GenRowMajor>>;

    int64_t BLK_M = KernelTraits::BLK_M;
    int64_t BLK_K = KernelTraits::BLK_K;

    size_t n_warpups = 100;
    GPU_Clock clock;

    cutlass::DeviceAllocation<scalar_t> data_A(M * K);
    cutlass::DeviceAllocation<scalar_t> data_B(N * K);
    cutlass::DeviceAllocation<scalar_t> data_C(M * N);
    cutlass::DeviceAllocation<ct::uint128_t> mask_data(M / BLK_M);

    auto A = ct::make_tensor(ct::make_gmem_ptr(data_A.get()),
                             ct::make_layout(ct::make_shape(M, K), LayoutA{}));
    auto B = ct::make_tensor(ct::make_gmem_ptr(data_B.get()),
                             ct::make_layout(ct::make_shape(N, K), LayoutB{}));
    auto C = ct::make_tensor(ct::make_gmem_ptr(data_C.get()),
                             ct::make_layout(ct::make_shape(M, N), ct::GenRowMajor{}));
    auto mask = ct::make_tensor(ct::make_gmem_ptr(mask_data.get()), ct::make_shape(M / BLK_M));

    for (size_t i = 0; i < n_warpups; i++) {
        auto [mask_host, mask_T, mask_table] = make_mask(M / BLK_M, K / BLK_K, 0.0);
        // std::cout << "Mask proportion: " << double(mask_table.size()) / (M / BLK_M * K / BLK_K)
        //           << std::endl;
        mask_data.copy_from_host(mask_host.data());
        matmul<KernelTraits>(A, B, C, mask);
    }

    clock.start();
    for (size_t i = 0; i < n_repeats; i++) {
        auto [mask_host, mask_T, mask_table] = make_mask(M / BLK_M, K / BLK_K, 0.0);
        mask_data.copy_from_host(mask_host.data());
        matmul<KernelTraits>(A, B, C, mask);
    }
    auto duration = clock.seconds();

    auto flops = 2.0 * M * N * K / duration * n_repeats;
    std::cout << "TFLOPs: " << flops / 1e12 << std::endl;

    return 0;
}
